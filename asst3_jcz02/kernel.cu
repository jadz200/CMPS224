#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define TILE_DIM 32

__global__ void mm_tiled_kernel(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    // TODO























}

void mm_gpu(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO
    floar *A_d, *B_d, *C_d;
    hipMalloc((void**) &A_d, M*K*sizeof(float));
    hipMalloc((void**) &B_d, K*N*sizeof(float));
    hipMalloc((void**) &C_d, M*N*sizeof(float));





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO
   





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO






    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO






    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO






    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

