#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

#define TILE_DIM 32

__global__ void mm_tiled_kernel(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    // TODO
    __shared__ float A_s [TILE_DIM][TILE_DIM];
    __shared__ float B_s [TILE_DIM][TILE_DIM];

    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
    
    float sum = 0.0f;

    for(unsigned int tile=0;tile<(K+TILE_DIM-1)/TILE_DIM;++tile){
	if(tile*TILE_DIM+threadIdx.x<K && row<M){
		A_s[threadIdx.y][threadIdx.x]= A[row*K + tile*TILE_DIM + threadIdx.x];
	}
	else{
		A_s[threadIdx.y][threadIdx.x]=0;
	}
	if(tile*TILE_DIM+threadIdx.y<K && col<N){
		B_s[threadIdx.y][threadIdx.x]= B[(tile*TILE_DIM + threadIdx.y)*N+col];
	}
	else{
		B_s[threadIdx.y][threadIdx.x]=0;
	}
	__syncthreads();
	for(unsigned int i = 0; i<TILE_DIM;++i){
			sum+=A_s[threadIdx.y][i]*B_s[i][threadIdx.x];
	}
	__syncthreads();
    }
    if (row<M && col < N){    
	C[row*N+col]=sum;
   }

}

void mm_gpu(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO
    float *A_d, *B_d, *C_d;
    hipMalloc((void**) &A_d, M*K*sizeof(float));
    hipMalloc((void**) &B_d, K*N*sizeof(float));
    hipMalloc((void**) &C_d, M*N*sizeof(float));

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO
    hipMemcpy(A_d, A, M*K*sizeof(float),hipMemcpyHostToDevice);  
    hipMemcpy(B_d, B, K*N*sizeof(float),hipMemcpyHostToDevice);  

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO
   dim3 numThreadsPerBlock(32,32);
   dim3 numBlocks((N+numThreadsPerBlock.x-1)/numThreadsPerBlock.x,(M+numThreadsPerBlock.y-1)/numThreadsPerBlock.y);

   mm_tiled_kernel<<<numBlocks, numThreadsPerBlock>>>(A_d,B_d,C_d,M,N,K);

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    hipMemcpy(C, C_d, M*N*sizeof(float),hipMemcpyDeviceToHost);   

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}
